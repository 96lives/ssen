#include "hip/hip_runtime.h"
/* Copyright (c) Chris Choy (chrischoy@ai.stanford.edu).
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 *
 * Please cite "4D Spatio-Temporal ConvNets: Minkowski Convolutional Neural
 * Networks", CVPR'19 (https://arxiv.org/abs/1904.08755) if you use any part
 * of the code.
 */
#ifndef GPU_BROADCAST
#define GPU_BROADCAST

#include "broadcast.cuh"
#include "math_functions.hpp"

template <class T> struct IsIntType { static const bool value = false; };

template <> struct IsIntType<int> { static const bool value = true; };

template <typename Dtype, typename Itype>
__global__ void
channelwise_addition(const int n, const int nchannel, const Dtype *d_glob_feat,
                     const Itype *d_sorted_map, Dtype *d_out_feat) {
  int row, ch_index;
  CUDA_KERNEL_LOOP(index, n) {
    ch_index = index % nchannel;
    row = d_sorted_map[index / nchannel];
    d_out_feat[index] += d_glob_feat[row * nchannel + ch_index];
  }
}

template <typename Dtype, typename Itype>
__global__ void channelwise_multiplication(const int n, const int nchannel,
                                           const Dtype *d_glob_feat,
                                           const Itype *d_sorted_out_map,
                                           Dtype *d_out_feat) {
  int row, ch_index;
  CUDA_KERNEL_LOOP(index, n) {
    ch_index = index % nchannel;
    row = d_sorted_out_map[index / nchannel];
    d_out_feat[index] *= d_glob_feat[row * nchannel + ch_index];
  }
}

template <typename Dtype, typename Itype>
__global__ void
channelwise_division(const int n, const int nchannel, const Dtype *d_glob_feat,
                     const Itype *d_sorted_out_map, Dtype *d_out_feat) {
  int row, ch_index;
  CUDA_KERNEL_LOOP(index, n) {
    ch_index = index % nchannel;
    row = d_sorted_out_map[index / nchannel];
    d_out_feat[index] /= d_glob_feat[row * nchannel + ch_index];
  }
}

template <typename Dtype>
__global__ void fill(const int n, Dtype *in_feat, Dtype val) {
  CUDA_KERNEL_LOOP(index, n) { in_feat[index] = val; }
}

template <typename Dtype, typename Itype>
void BroadcastForwardKernelGPU(
    const Dtype *d_in_feat, int in_nrows, const Dtype *d_in_feat_global,
    int in_nrows_global, Dtype *d_out_feat, int nchannel, int op,
    const std::vector<std::vector<Itype>> &sorted_in_maps,
    const std::vector<std::vector<Itype>> &sorted_out_maps, Itype *d_scr,
    hipsparseHandle_t cushandle, hipStream_t stream) {
  Itype *d_sorted_out_map;

  if (sorted_in_maps.size() != 1)
    throw std::invalid_argument("InOut map must have one kernel for Broadcast");

  if (sorted_in_maps[0].size() != in_nrows) {
    std::cout << "sorted_in_map[0].size(): " << sorted_in_maps[0].size()
              << ", in_nrows: " << in_nrows << std::endl;
    throw std::invalid_argument("Invalid in_map");
  }

  // CUDA_CHECK(hipMalloc((void **)&d_sorted_out_map,
  //                       sorted_out_maps[0].size() * sizeof(Itype)));
  d_sorted_out_map = d_scr;
  // Copy all in_feat to out_feat
  CUDA_CHECK(hipMemcpy(d_out_feat, d_in_feat,
                        sizeof(Dtype) * nchannel * in_nrows,
                        hipMemcpyDeviceToDevice));
  CUDA_CHECK(hipMemcpy(d_sorted_out_map, sorted_out_maps[0].data(),
                        sizeof(Itype) * sorted_out_maps[0].size(),
                        hipMemcpyHostToDevice));

  // To speed up, put switch outside for loops
  switch (op) {
  case 0: // +
    channelwise_addition<Dtype, Itype>
        <<<GET_BLOCKS(in_nrows * nchannel), CUDA_NUM_THREADS, 0, stream>>>(
            nchannel * in_nrows, nchannel, d_in_feat_global, d_sorted_out_map,
            d_out_feat);
    break;
  case 1: // *
    channelwise_multiplication<Dtype, Itype>
        <<<GET_BLOCKS(in_nrows * nchannel), CUDA_NUM_THREADS, 0, stream>>>(
            nchannel * in_nrows, nchannel, d_in_feat_global, d_sorted_out_map,
            d_out_feat);
    break;
  default:
    throw std::invalid_argument(Formatter() << "Operation not supported: "
                                            << std::to_string(op));
  }
  // hipFree(d_sorted_out_map);
}

template void BroadcastForwardKernelGPU<float, int32_t>(
    const float *d_in_feat, int in_nrows, const float *d_in_feat_global,
    int in_nrows_global, float *d_out_feat, int nchannel, int op,
    const std::vector<std::vector<int32_t>> &sorted_in_map,
    const std::vector<std::vector<int32_t>> &sorted_out_map, int32_t *d_scr,
    hipsparseHandle_t cuhandle, hipStream_t stream);

template void BroadcastForwardKernelGPU<double, int32_t>(
    const double *d_in_feat, int in_nrows, const double *d_in_feat_global,
    int in_nrows_global, double *d_out_feat, int nchannel, int op,
    const std::vector<std::vector<int32_t>> &sorted_in_map,
    const std::vector<std::vector<int32_t>> &sorted_out_map, int32_t *d_scr,
    hipsparseHandle_t cuhandle, hipStream_t stream);

template <typename Dtype, typename Itype>
void BroadcastBackwardKernelGPU(
    const Dtype *d_in_feat, Dtype *d_grad_in_feat, int in_nrows,
    const Dtype *d_in_feat_global, Dtype *d_grad_in_feat_global,
    int in_nrows_global, const Dtype *d_grad_out_feat, int nchannel, int op,
    const std::vector<std::vector<Itype>> &sorted_in_maps,
    const std::vector<std::vector<Itype>> &sorted_out_maps, Itype *d_scr,
    Dtype *d_dscr, hipsparseHandle_t cushandle, hipStream_t stream) {
  Itype *d_sorted_in_map, *d_sorted_out_map, *d_csr_row;
  Dtype *d_dtype, *d_csr_val, *d_tmp_grad_in_feat_global, *d_tmp_grad_in_feat;
  hipsparseMatDescr_t descr = 0;
  const Dtype alpha = 1;
  const Dtype beta = 0;
  int nnz = in_nrows;

  if (!IsIntType<Itype>::value)
    throw std::invalid_argument("Not implemented"); // Due to hipsparseXcoo2csr

  if (sorted_in_maps.size() != 1)
    throw std::invalid_argument("InOut map must have one kernel for Broadcast");

  if (sorted_in_maps[0].size() != in_nrows)
    throw std::invalid_argument("Invalid in_map");

  // Malloc d_sorted_in_map, d_sorted_out_map, d_csr_row
  // THRUST_CHECK(d_csr_row.resize(in_nrows_global + 1));
  // CSR returns n_row + 1
  // CUDA_CHECK(hipMalloc((void **)&d_sorted_in_map,
  //                       (sorted_in_maps[0].size() + sorted_out_maps[0].size()
  //                       + in_nrows_global + 1) * sizeof(Itype)));
  d_sorted_in_map = d_scr;
  d_sorted_out_map = d_sorted_in_map + sorted_in_maps[0].size();
  d_csr_row = d_sorted_out_map + sorted_out_maps[0].size();

  // GPUMemoryManager<Dtype> dmem((nnz + (in_nrows + in_nrows_global) *
  // nchannel)); CUDA_CHECK(hipMalloc((void **)&d_dtype,
  //                       (nnz + (in_nrows + in_nrows_global) * nchannel) *
  //                           sizeof(Dtype)));
  // d_dtype =
  //     (Dtype *)(d_scr + sorted_in_maps[0].size() + sorted_out_maps[0].size()
  //     + in_nrows_global + 1);
  d_dtype = d_dscr;
  d_tmp_grad_in_feat_global = d_dtype;
  d_tmp_grad_in_feat = d_tmp_grad_in_feat_global + in_nrows_global * nchannel;
  d_csr_val = d_tmp_grad_in_feat + in_nrows * nchannel;

  // COO cols
  // THRUST_CHECK(d_sorted_in_map = sorted_in_map[0]);    // COO cols
  CUDA_CHECK(hipMemcpy(d_sorted_in_map, sorted_in_maps[0].data(),
                        sizeof(Itype) * sorted_in_maps[0].size(),
                        hipMemcpyHostToDevice));
  // COO rows
  // THRUST_CHECK(d_sorted_out_map = sorted_out_map[0]);  // COO rows
  CUDA_CHECK(hipMemcpy(d_sorted_out_map, sorted_out_maps[0].data(),
                        sizeof(Itype) * sorted_out_maps[0].size(),
                        hipMemcpyHostToDevice));

  // thrust::fill(d_csr_val.begin(), d_csr_val.end(), 1);
  fill<Dtype><<<GET_BLOCKS(in_nrows), CUDA_NUM_THREADS, 0, stream>>>(
      nnz, d_csr_val, (Dtype)1.);

  CUSPARSE_CHECK(hipsparseCreateMatDescr(&descr));
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

  // Sort COO first
  sort_coo_gpu(cushandle, in_nrows_global, in_nrows, nnz, d_sorted_out_map,
               d_sorted_in_map);
  // For CRS, sort row and col inds by row major.
  CUSPARSE_CHECK(hipsparseXcoo2csr(cushandle, d_sorted_out_map, nnz,
                                  in_nrows_global, d_csr_row,
                                  HIPSPARSE_INDEX_BASE_ZERO));

  // To speed up, put switch outside for loops
  switch (op) {
  case 0: // +
    // For grad_in_feat, copy all grad_out_feat to grad_in_feat
    CUDA_CHECK(hipMemcpy(d_grad_in_feat, d_grad_out_feat,
                          sizeof(Dtype) * nchannel * in_nrows,
                          hipMemcpyDeviceToDevice));
    // For grad_in_feat_glob, add all grad_out_feat
    CUSPARSE_CHECK(
        cusparse_csrmm<Dtype>(cushandle,
                              HIPSPARSE_OPERATION_NON_TRANSPOSE, // op(A)
                              HIPSPARSE_OPERATION_TRANSPOSE,     // op(B)
                              in_nrows_global,                  // M
                              nchannel,                         // N
                              in_nrows,                         // K
                              nnz, &alpha, descr,
                              d_csr_val,       // val
                              d_csr_row,       // row
                              d_sorted_in_map, // col
                              d_grad_out_feat, // B
                              nchannel,        // ldb
                              &beta,
                              d_tmp_grad_in_feat_global, // C
                              in_nrows_global            // ldc
                              ));

    col2row_major<Dtype>(in_nrows_global, nchannel, d_tmp_grad_in_feat_global,
                         d_grad_in_feat_global, stream);
    break;
  case 1: // *
    // First, for grad_in_feat
    // Copy in_feat_global to tmp, then multiply the tmp with grad_out_feat
    row2col_major<Dtype>(in_nrows_global, nchannel, d_in_feat_global,
                         d_tmp_grad_in_feat_global, stream);
    CUSPARSE_CHECK(
        cusparse_csrmm<Dtype>(cushandle,
                              HIPSPARSE_OPERATION_TRANSPOSE,     // op(A)
                              HIPSPARSE_OPERATION_NON_TRANSPOSE, // op(B)
                              in_nrows_global,                  // M
                              nchannel,                         // N
                              in_nrows,                         // K
                              nnz, &alpha, descr,
                              d_csr_val,                 // val
                              d_csr_row,                 // row
                              d_sorted_in_map,           // col
                              d_tmp_grad_in_feat_global, // B
                              in_nrows_global,           // ldb
                              &beta,
                              d_tmp_grad_in_feat, // C
                              in_nrows            // ldc
                              ));
    col2row_major<Dtype>(in_nrows, nchannel, d_tmp_grad_in_feat, d_grad_in_feat,
                         stream);
    gpu_multiplication<Dtype>(nchannel * in_nrows, d_grad_out_feat,
                              d_grad_in_feat, d_grad_in_feat, stream);

    // Second, for grad_in_feat_global, copy in_feat to tmp,
    CUDA_CHECK(hipMemcpy(d_tmp_grad_in_feat, d_grad_out_feat,
                          sizeof(Dtype) * nchannel * in_nrows,
                          hipMemcpyDeviceToDevice));
    gpu_multiplication<Dtype>(nchannel * in_nrows, d_in_feat,
                              d_tmp_grad_in_feat, d_tmp_grad_in_feat, stream);
    CUSPARSE_CHECK(
        cusparse_csrmm<Dtype>(cushandle,
                              HIPSPARSE_OPERATION_NON_TRANSPOSE, // op(A)
                              HIPSPARSE_OPERATION_TRANSPOSE,     // op(B)
                              in_nrows_global,                  // M
                              nchannel,                         // N
                              in_nrows,                         // K
                              nnz, &alpha, descr,
                              d_csr_val,          // val
                              d_csr_row,          // row
                              d_sorted_in_map,    // col
                              d_tmp_grad_in_feat, // B
                              nchannel,           // ldb
                              &beta,
                              d_tmp_grad_in_feat_global, // C
                              in_nrows_global            // ldc
                              ));
    col2row_major<Dtype>(in_nrows_global, nchannel, d_tmp_grad_in_feat_global,
                         d_grad_in_feat_global, stream);
    break;
  default:
    throw std::invalid_argument(Formatter() << "Operation not supported: "
                                            << std::to_string(op));
  }

  CUSPARSE_CHECK(hipsparseDestroyMatDescr(descr));

  // hipFree(d_sorted_in_map);
  // hipFree(d_dtype);
}

template void BroadcastBackwardKernelGPU<float, int32_t>(
    const float *d_in_feat, float *d_grad_in_feat, int in_nrows,
    const float *d_in_feat_global, float *d_grad_in_feat_global,
    int in_nrows_global, const float *d_grad_out_feat, int nchannel, int op,
    const std::vector<std::vector<int32_t>> &sorted_in_map,
    const std::vector<std::vector<int32_t>> &sorted_out_map, int32_t *d_scr,
    float *d_dscr, hipsparseHandle_t cushandle, hipStream_t stream);

template void BroadcastBackwardKernelGPU<double, int32_t>(
    const double *d_in_feat, double *d_grad_in_feat, int in_nrows,
    const double *d_in_feat_global, double *d_grad_in_feat_global,
    int in_nrows_global, const double *d_grad_out_feat, int nchannel, int op,
    const std::vector<std::vector<int32_t>> &sorted_in_map,
    const std::vector<std::vector<int32_t>> &sorted_out_map, int32_t *d_scr,
    double *d_dscr, hipsparseHandle_t cushandle, hipStream_t stream);
#endif
