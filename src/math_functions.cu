#include "hip/hip_runtime.h"
/*  Copyright (c) Chris Choy (chrischoy@ai.stanford.edu).
 *
 *  Permission is hereby granted, free of charge, to any person obtaining a copy of
 *  this software and associated documentation files (the "Software"), to deal in
 *  the Software without restriction, including without limitation the rights to
 *  use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies
 *  of the Software, and to permit persons to whom the Software is furnished to do
 *  so, subject to the following conditions:
 *
 *  The above copyright notice and this permission notice shall be included in all
 *  copies or substantial portions of the Software.
 *
 *  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 *  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 *  AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 *  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 *  OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 *  SOFTWARE.
 *
 *  Please cite "4D Spatio-Temporal ConvNets: Minkowski Convolutional Neural
 *  Networks", CVPR'19 (https://arxiv.org/abs/1904.08755) if you use any part
 *  of the code.
 */
#include "math_functions.hpp"

// CUBLAS, CUSPARSE assume all dense matrices to be col major
template <>
void gpu_gemm<float>(hipblasHandle_t handle, const CBLAS_TRANSPOSE TransA,
                     const CBLAS_TRANSPOSE TransB, const int M, const int N,
                     const int K, const float alpha, const float *A,
                     const float *B, const float beta, float *C) {
  // Note that cublas follows (column-major) fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(handle, cuTransB, cuTransA, N, M, K, &alpha, B, ldb,
                           A, lda, &beta, C, N));
}

template <>
void gpu_gemm<double>(hipblasHandle_t handle, const CBLAS_TRANSPOSE TransA,
                      const CBLAS_TRANSPOSE TransB, const int M, const int N,
                      const int K, const double alpha, const double *A,
                      const double *B, const double beta, double *C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(handle, cuTransB, cuTransA, N, M, K, &alpha, B, ldb,
                           A, lda, &beta, C, N));
}

// CUBLAS, CUSPARSE assume all dense matrices to be col major
// If op(B)=B, cusparse<t>csrmm2() is the same as cusparse<t>csrmm();
// otherwise, only op(A)=A is supported and the matrix type must be
// HIPSPARSE_MATRIX_TYPE_GENERAL.
// M: # row of A
// N: # col of op(B) or C
// K: # col of A
template <>
hipsparseStatus_t
cusparse_csrmv<float>(hipsparseHandle_t handle, hipsparseOperation_t transA,
                      int m, int n, int nnz, const float *alpha,
                      const hipsparseMatDescr_t descrA, const float *csrValA,
                      const int *csrRowPtrA, const int *csrColIndA,
                      const float *x, const float *beta, float *y) {
  return hipsparseScsrmv(handle, transA, m, n, nnz, alpha, descrA, csrValA,
                        csrRowPtrA, csrColIndA, x, beta, y);
};

template <>
hipsparseStatus_t
cusparse_csrmv<double>(hipsparseHandle_t handle, hipsparseOperation_t transA,
                       int m, int n, int nnz, const double *alpha,
                       const hipsparseMatDescr_t descrA, const double *csrValA,
                       const int *csrRowPtrA, const int *csrColIndA,
                       const double *x, const double *beta, double *y) {
  return hipsparseDcsrmv(handle, transA, m, n, nnz, alpha, descrA, csrValA,
                        csrRowPtrA, csrColIndA, x, beta, y);
};

template <>
hipsparseStatus_t
cusparse_csrmm<float>(hipsparseHandle_t handle, hipsparseOperation_t transA,
                      hipsparseOperation_t transB, int m, int n, int k, int nnz,
                      const float *alpha, const hipsparseMatDescr_t descrA,
                      const float *csrValA, const int *csrRowPtrA,
                      const int *csrColIndA, const float *B, int ldb,
                      const float *beta, float *C, int ldc) {
  return hipsparseScsrmm2(handle, transA, transB, m, n, k, nnz, alpha, descrA,
                         csrValA, csrRowPtrA, csrColIndA, B, ldb, beta, C, ldc);
}

template <>
hipsparseStatus_t
cusparse_csrmm<double>(hipsparseHandle_t handle, hipsparseOperation_t transA,
                       hipsparseOperation_t transB, int m, int n, int k, int nnz,
                       const double *alpha, const hipsparseMatDescr_t descrA,
                       const double *csrValA, const int *csrRowPtrA,
                       const int *csrColIndA, const double *B, int ldb,
                       const double *beta, double *C, int ldc) {
  return hipsparseDcsrmm2(handle, transA, transB, m, n, k, nnz, alpha, descrA,
                         csrValA, csrRowPtrA, csrColIndA, B, ldb, beta, C, ldc);
}

template <typename Dtype>
__global__ void addition_kernel(const int n, const Dtype *a, const Dtype *b,
                                Dtype *y) {
  CUDA_KERNEL_LOOP(index, n) { y[index] = a[index] + b[index]; }
}

template <typename Dtype>
__global__ void multiplication_kernel(const int n, const Dtype *a,
                                      const Dtype *b, Dtype *y) {
  CUDA_KERNEL_LOOP(index, n) { y[index] = a[index] * b[index]; }
}

template <typename Dtype>
void gpu_addition(const int N, const Dtype *a, const Dtype *b, Dtype *y,
                  hipStream_t stream) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  addition_kernel<Dtype>
      <<<GET_BLOCKS(N), CUDA_NUM_THREADS, 0, stream>>>(N, a, b, y);
}

template void gpu_addition<float>(const int N, const float *a, const float *b,
                                  float *y, hipStream_t stream);

template void gpu_addition<double>(const int N, const double *a,
                                   const double *b, double *y,
                                   hipStream_t stream);

template <typename Dtype>
void gpu_multiplication(const int N, const Dtype *a, const Dtype *b, Dtype *y,
                        hipStream_t stream) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  multiplication_kernel<Dtype>
      <<<GET_BLOCKS(N), CUDA_NUM_THREADS, 0, stream>>>(N, a, b, y);
}

template void gpu_multiplication<float>(const int N, const float *a,
                                        const float *b, float *y,
                                        hipStream_t stream);

template void gpu_multiplication<double>(const int N, const double *a,
                                         const double *b, double *y,
                                         hipStream_t stream);

template <typename Dtype>
__global__ void col2row_major_kernel(const int n, const int nrows,
                                     const int ncols, const Dtype *colA,
                                     Dtype *rowA) {
  int i, j;
  CUDA_KERNEL_LOOP(index, n) {
    i = index % nrows;
    j = index / nrows;
    rowA[i * ncols + j] = colA[index];
  }
}

template <typename Dtype>
void col2row_major(const int nrows, const int ncols, const Dtype *colA,
                   Dtype *rowA, hipStream_t stream) {
  col2row_major_kernel<Dtype>
      <<<GET_BLOCKS(nrows * ncols), CUDA_NUM_THREADS, 0, stream>>>(
          nrows * ncols, nrows, ncols, colA, rowA);
}

template void col2row_major<float>(const int nrows, const int ncols,
                                   const float *colA, float *rowA,
                                   hipStream_t stream);

template void col2row_major<double>(const int nrows, const int ncols,
                                    const double *colA, double *rowA,
                                    hipStream_t stream);

template <typename Dtype>
__global__ void row2col_major_kernel(const int n, const int nrows,
                                     const int ncols, const Dtype *rowA,
                                     Dtype *colA) {
  int i, j;
  CUDA_KERNEL_LOOP(index, n) {
    i = index / ncols;
    j = index % ncols;
    colA[i + j * nrows] = rowA[index];
  }
}

template <typename Dtype>
void row2col_major(const int nrows, const int ncols, const Dtype *colA,
                   Dtype *rowA, hipStream_t stream) {
  row2col_major_kernel<Dtype>
      <<<GET_BLOCKS(nrows * ncols), CUDA_NUM_THREADS, 0, stream>>>(
          nrows * ncols, nrows, ncols, colA, rowA);
}

template void row2col_major<float>(const int nrows, const int ncols,
                                   const float *colA, float *rowA,
                                   hipStream_t stream);

template void row2col_major<double>(const int nrows, const int ncols,
                                    const double *colA, double *rowA,
                                    hipStream_t stream);

// Sort (row, col) pairs row-major order.
void sort_coo_gpu(hipsparseHandle_t handle, const int m, const int n,
                  const int nnz, int *d_coo_row, int *d_coo_col) {
  size_t pBufferSizeInBytes = 0;
  void *pBuffer = NULL;
  int *P = NULL;

  // step 1: allocate buffer
  CUSPARSE_CHECK(hipsparseXcoosort_bufferSizeExt(
      handle, m, n, nnz, d_coo_row, d_coo_col, &pBufferSizeInBytes));
  CUDA_CHECK(hipMalloc(&pBuffer, sizeof(char) * pBufferSizeInBytes));
  // step 2: setup permutation vector P to identity
  CUDA_CHECK(hipMalloc((void **)&P, sizeof(int) * nnz));
  CUSPARSE_CHECK(hipsparseCreateIdentityPermutation(handle, nnz, P));
  // step 3: sort COO
  CUSPARSE_CHECK(hipsparseXcoosortByRow(handle, m, n, nnz, d_coo_row, d_coo_col,
                                       P, pBuffer));
  hipFree(pBuffer);
  hipFree(P);
}
